#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "convolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    binary[i] = (x[i] >= 0) ? 1 : -1;
}

void binarize_gpu(float *x, int n, float *binary)
{
    binarize_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, binary);
    check_error(hipPeekAtLastError());
}

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < n; ++i){
        mean += fabsf(input[i*size + s]);
    }
    mean = mean / n;
    for(i = 0; i < n; ++i){
        binary[i*size + s] = (input[i*size + s] > 0) ? mean : -mean;
    }
}

void binarize_input_gpu(float *input, int n, int size, float *binary)
{
    binarize_input_kernel<<<cuda_gridsize(size), BLOCK>>>(input, n, size, binary);
    check_error(hipPeekAtLastError());
}


__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < size; ++i){
        mean += fabsf(weights[f*size + i]);
    }
    mean = mean / size;
    for(i = 0; i < size; ++i){
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
        //binary[f*size + i] = weights[f*size + i];
    }
}

void binarize_weights_gpu(float *weights, int n, int size, float *binary)
{
    binarize_weights_kernel<<<cuda_gridsize(n), BLOCK>>>(weights, n, size, binary);
    check_error(hipPeekAtLastError());
}

__global__ void cuda_f32_to_f16(float* input_f32, size_t size, half *output_f16)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) output_f16[idx] = __float2half(input_f32[idx]);
    //if (idx < size) *((unsigned short *)output_f16 + idx) = __float2half(input_f32[idx]);
}

void cuda_convert_f32_to_f16(float* input_f32, size_t size, float *output_f16) {
    cuda_f32_to_f16 <<< size / BLOCK + 1, BLOCK, 0, get_cuda_stream() >>> (input_f32, size, (half *)output_f16);
    check_error(hipPeekAtLastError());
}

__global__ void cuda_f16_to_f32(half* input_f16, size_t size, float *output_f32)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) output_f32[idx] = __half2float(input_f16[idx]);
    //if (idx < size) output_f32[idx] = __half2float(*((unsigned short *)input_f16 + idx));
}

void cuda_convert_f16_to_f32(float* input_f16, size_t size, float *output_f32) {
    cuda_f16_to_f32 <<< size / BLOCK + 1, BLOCK, 0, get_cuda_stream() >>> ((half *)input_f16, size, output_f32);
    check_error(hipPeekAtLastError());
}

half *cuda_make_f16_from_f32_array(float *src, size_t n)
{
    half *dst16;
    size_t size = sizeof(half)*n;
    check_error(hipMalloc((void **)&dst16, size));
    if (src) {
        cuda_convert_f32_to_f16(src, n, (float *)dst16);
    }
    if (!dst16) error("Cuda malloc failed\n");
    return dst16;
}

void forward_convolutional_layer_gpu(convolutional_layer l, network net)
{
    fill_gpu(l.outputs*l.batch, 0, l.output_gpu, 1);
    if(l.binary){
        binarize_weights_gpu(l.weights_gpu, l.n, l.c/l.groups*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
    }

    if(l.xnor){
        binarize_weights_gpu(l.weights_gpu, l.n, l.c/l.groups*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
        binarize_gpu(net.input_gpu, l.c*l.h*l.w*l.batch, l.binary_input_gpu);
        net.input_gpu = l.binary_input_gpu;
    }

#ifdef CUDNN
    float one = 1;    // alpha[0], beta[0] is float for HALF and FLOAT
    float alpha = 1, beta = 0; 

#ifdef CUDNN_HALF
    // Note: For improved performance it is advised to use beta[0] = 0.0. 
    // For Tensor Core: hipdnnSetConvolutionMathType() where hipdnnMathType_t mathType = HIPDNN_TENSOR_OP_MATH;
    // 1. or HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM and use HIPDNN_DATA_HALF
    // 2. or HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED
    // More: http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#tensor_ops

    const size_t input16_size = l.batch*l.c*l.w*l.h;
    const size_t output16_size = l.batch*l.out_c*l.out_h*l.out_w;

    if (*net.max_input16_size < input16_size) {
        //printf("\n input16_size: cur = %zu \t max = %zu \n", input16_size, *net.max_input16_size);
        *net.max_input16_size = input16_size;
        if (*net.input16_gpu) cuda_free(*net.input16_gpu);
        *net.input16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *net.max_input16_size);
    }
    float *input16 = *net.input16_gpu;

    if (*net.max_output16_size < output16_size) {
        *net.max_output16_size = output16_size;
        if (*net.output16_gpu) cuda_free(*net.output16_gpu);
        *net.output16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *net.max_output16_size);
    }
    float *output16 = *net.output16_gpu;

    cuda_convert_f32_to_f16(net.input_gpu, input16_size, input16);

    hipdnnConvolutionForward(cudnn_handle(),
        &alpha,
        l.srcTensorDesc,
        input16,
        l.weightDesc,
        l.weights_gpu16,
        l.convDesc,
        l.fw_algo,
        net.workspace,
        l.workspace_size,
        &beta,
        l.dstTensorDesc,
        output16);

    if (l.batch_normalize){
        if (net.train){ // Training
            copy_gpu(l.outputs*l.batch / 2, output16, 1, l.x_gpu, 1);
            float zero = 0;
            // Batch-normalization can still take FP16 inputs and outputs, saving half the bandwidth
            // compared to FP32, it is just that the statistics and value adjustment should be done in FP32.
            hipdnnBatchNormalizationForwardTraining(cudnn_handle(),
                HIPDNN_BATCHNORM_SPATIAL,
                &one,
                &zero,
                l.normDstTensorDescF16,
                l.x_gpu,            // input
                l.normDstTensorDescF16,
                output16,            // output
                l.normTensorDesc,
                l.scales_gpu,
                l.biases_gpu,
                .01,
                l.rolling_mean_gpu,        // output (should be FP32)
                l.rolling_variance_gpu,    // output (should be FP32)
                .00001,
                l.mean_gpu,            // output (should be FP32)
                l.variance_gpu);    // output (should be FP32)

            cuda_convert_f16_to_f32(output16, output16_size, l.output_gpu);
            //forward_batchnorm_layer_gpu(l, net);
        }
        else{ // Detection
            cuda_convert_f16_to_f32(output16, output16_size, l.output_gpu);
            normalize_gpu(l.output_gpu, l.rolling_mean_gpu, l.rolling_variance_gpu, l.batch, l.out_c, l.out_h*l.out_w);
            scale_bias_gpu(l.output_gpu, l.scales_gpu, l.batch, l.out_c, l.out_h*l.out_w);
            add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.out_c, l.out_w*l.out_h);
        }
    }
    else{ // BIAS only
        cuda_convert_f16_to_f32(output16, output16_size, l.output_gpu);
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }

#else

    hipdnnConvolutionForward(cudnn_handle(),
                &one,
                l.srcTensorDesc,
                net.input_gpu,
                l.weightDesc,
                l.weights_gpu,
                l.convDesc,
                l.fw_algo,
                net.workspace,
                l.workspace_size,
                &one,
                l.dstTensorDesc,
                l.output_gpu);
#endif    // CUDNN_HALF


#else
    int i, j;
    int m = l.n/l.groups;
    int k = l.size*l.size*l.c/l.groups;
    int n = l.out_w*l.out_h;
    for(i = 0; i < l.batch; ++i){
        for(j = 0; j < l.groups; ++j){
            float *a = l.weights_gpu + j*l.nweights/l.groups;
            float *b = net.workspace;
            float *c = l.output_gpu + (i*l.groups + j)*n*m;
            float *im = net.input_gpu + (i*l.groups + j)*l.c/l.groups*l.h*l.w;

            if (l.size == 1){
                b = im;
            } else {
                im2col_gpu(im, l.c/l.groups, l.h, l.w, l.size, l.stride, l.pad, b);
            }
            gemm_gpu(0,0,m,n,k,1,a,k,b,n,1,c,n);
        }
    }
#endif

#ifndef CUDNN_HALF
    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, net);
    } else {
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
#endif // no CUDNN_HALF

    activate_array_gpu(l.output_gpu, l.outputs*l.batch, l.activation);
    //if(l.dot > 0) dot_error_gpu(l);
    if(l.binary || l.xnor) swap_binary(&l);
}

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -(size/2.f);
    int h_offset = -(size/2.f);

    int out_index = j + w*(i + h*(k + c*b));
    int l, m;
    for(l = 0; l < size; ++l){
        for(m = 0; m < size; ++m){
            int cur_h = h_offset + i + l;
            int cur_w = w_offset + j + m;
            int index = cur_w + w*(cur_h + h*(k + b*c));
            int valid = (cur_h >= 0 && cur_h < h &&
                    cur_w >= 0 && cur_w < w);
            delta[out_index] += valid ? rate*(x[index] - x[out_index]) : 0;
        }
    }
}

extern "C" void smooth_layer(layer l, int size, float rate)
{
    int h = l.out_h;
    int w = l.out_w;
    int c = l.out_c;

    size_t n = h*w*c*l.batch;

    smooth_kernel<<<cuda_gridsize(n), BLOCK>>>(l.output_gpu, n, l.w, l.h, l.c, size, rate, l.delta_gpu);
    check_error(hipPeekAtLastError());
}

void backward_convolutional_layer_gpu(convolutional_layer l, network net)
{
    if(l.smooth){
        smooth_layer(l, 5, l.smooth);
    }
    //constrain_gpu(l.outputs*l.batch, 1, l.delta_gpu, 1);
    gradient_array_gpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);


#ifndef CUDNN_HALF
    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, net);
    } else {
        backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
#endif // no CUDNN_HALF
    float *original_input = net.input_gpu;

    if(l.xnor) net.input_gpu = l.binary_input_gpu;
#ifdef CUDNN
    float one = 1;
    float alpha = 1, beta = 0;

#ifdef CUDNN_HALF
    const size_t input16_size = l.batch*l.c*l.w*l.h;
    const size_t delta16_size = l.batch*l.n*l.out_w*l.out_h;
    
    if (*net.max_input16_size < input16_size) {
        *net.max_input16_size = input16_size;
        if(*net.input16_gpu) cuda_free(*net.input16_gpu);
        *net.input16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *net.max_input16_size);
    }
    float *input16 = *net.input16_gpu;

    if (*net.max_output16_size < delta16_size) {
        *net.max_output16_size = delta16_size;
        if(*net.output16_gpu) cuda_free(*net.output16_gpu);
        *net.output16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *net.max_output16_size);
    }
    float *delta16 = *net.output16_gpu;

    cuda_convert_f32_to_f16(net.input_gpu, input16_size, input16);
    cuda_convert_f32_to_f16(l.delta_gpu, delta16_size, delta16);

    if (l.batch_normalize) {
        float one = 1;
        float zero = 0;
        hipdnnBatchNormalizationBackward(cudnn_handle(),
            HIPDNN_BATCHNORM_SPATIAL,
            &one,
            &zero,
            &one,
            &one,
            l.normDstTensorDescF16,
            l.x_gpu,                // input
            l.normDstTensorDescF16,
            delta16,                // input
            l.normDstTensorDescF16,
            l.x_norm_gpu,            // output
            l.normTensorDesc,
            l.scales_gpu,            // output (should be FP32)
            l.scale_updates_gpu,    // output (should be FP32)
            l.bias_updates_gpu,        // output (should be FP32)
            .00001,
            l.mean_gpu,                // input (should be FP32)
            l.variance_gpu);        // input (should be FP32)
        copy_gpu(l.outputs*l.batch / 2, l.x_norm_gpu, 1, delta16, 1);
    }
    else{
        backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    }

    // convert input: net.input (x), l.delta_gpu (y) from fp32 to fp16
    // get output: l.weight_updates_gpu (dw) and convert it to fp32 (ONLY if it is fp16)

    // calculate conv weight updates
    // Already: l.weight_updates_gpu = (l.weight_updates_gpu - l.weight*decay*batch*subdivision)*momentum
    //   so we should copy f32 to f16, or compute: f16=(w_up - w*d*b*s)*m
    cuda_convert_f32_to_f16(l.weight_updates_gpu, l.c*l.n*l.size*l.size, l.weight_updates_gpu16);

    hipdnnConvolutionBackwardFilter(cudnn_handle(),
        &one,
        l.srcTensorDesc,
        input16, //net.input,
        l.ddstTensorDesc,
        delta16, //l.delta_gpu,
        l.convDesc,
        l.bf_algo,
        net.workspace,
        l.workspace_size,
        &one,
        l.dweightDesc,
        l.weight_updates_gpu16);    // l.weight_updates_gpu);

    cuda_convert_f16_to_f32(l.weight_updates_gpu16, l.c*l.n*l.size*l.size, l.weight_updates_gpu);

    if (net.delta_gpu) {
        if (l.binary || l.xnor) swap_binary(&l);

        // http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#hipdnnConvolutionBackwardData
        // calculate delta for the next layer
        // convert input: l.weights_gpu (w), l.delta_gpu (dy) from fp32 to fp16
        // get output: net.delta (dx) and convert it to fp32 (ONLY if it is fp16)
        hipdnnConvolutionBackwardData(cudnn_handle(),
            &alpha,
            l.weightDesc,
            l.weights_gpu16, //l.weights_gpu,
            l.ddstTensorDesc,
            delta16, //l.delta_gpu,
            l.convDesc,
            l.bd_algo,
            net.workspace,
            l.workspace_size,
            &beta,
            l.dsrcTensorDesc,
            input16);    // net.delta_gpu);

        cuda_convert_f16_to_f32(input16, input16_size, net.delta_gpu);

        if (l.binary || l.xnor) swap_binary(&l);
        if (l.xnor) gradient_array_gpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, net.delta_gpu);
    }
#else    // CUDNN_HALF

    // calculate conv weight updates
    // if used: beta=1 then loss decreases faster
    hipdnnConvolutionBackwardFilter(cudnn_handle(),
            &one,
            l.srcTensorDesc,
            net.input_gpu,
            l.ddstTensorDesc,
            l.delta_gpu,
            l.convDesc,
            l.bf_algo,
            net.workspace,
            l.workspace_size,
            &one,
            l.dweightDesc,
            l.weight_updates_gpu);

    if(net.delta_gpu){
        if(l.binary || l.xnor) swap_binary(&l);
        // http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#hipdnnConvolutionBackwardData
        // calculate delta for the next layer
        hipdnnConvolutionBackwardData(cudnn_handle(),
                &one,
                l.weightDesc,
                l.weights_gpu,
                l.ddstTensorDesc,
                l.delta_gpu,
                l.convDesc,
                l.bd_algo,
                net.workspace,
                l.workspace_size,
                &one,
                l.dsrcTensorDesc,
                net.delta_gpu);
        if(l.binary || l.xnor) swap_binary(&l);
        if(l.xnor) gradient_array_gpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, net.delta_gpu);
    }

#endif    // CUDNN_HALF

#else    // CUDNN
    int m = l.n/l.groups;
    int n = l.size*l.size*l.c/l.groups;
    int k = l.out_w*l.out_h;

    int i, j;
    for(i = 0; i < l.batch; ++i){
        for(j = 0; j < l.groups; ++j){
            float *a = l.delta_gpu + (i*l.groups + j)*m*k;
            float *b = net.workspace;
            float *c = l.weight_updates_gpu + j*l.nweights/l.groups;

            float *im  = net.input_gpu+(i*l.groups + j)*l.c/l.groups*l.h*l.w;
            float *imd = net.delta_gpu+(i*l.groups + j)*l.c/l.groups*l.h*l.w;

            im2col_gpu(im, l.c/l.groups, l.h, l.w, l.size, l.stride, l.pad, b);
            gemm_gpu(0,1,m,n,k,1,a,k,b,k,1,c,n);

            if (net.delta_gpu) {
                if (l.binary || l.xnor) swap_binary(&l);
                a = l.weights_gpu + j*l.nweights/l.groups;
                b = l.delta_gpu + (i*l.groups + j)*m*k;
                c = net.workspace;
                if (l.size == 1) {
                    c = imd;
                }

                gemm_gpu(1,0,n,k,m,1,a,n,b,k,0,c,k);

                if (l.size != 1) {
                    col2im_gpu(net.workspace, l.c/l.groups, l.h, l.w, l.size, l.stride, l.pad, imd);
                }
                if(l.binary || l.xnor) {
                    swap_binary(&l);
                }
            }
            if(l.xnor) gradient_array_gpu(original_input + i*l.c*l.h*l.w, l.c*l.h*l.w, HARDTAN, net.delta_gpu + i*l.c*l.h*l.w);
        }
    }
#endif
}

void pull_convolutional_layer(layer l)
{
    cuda_pull_array(l.weights_gpu, l.weights, l.nweights);
    cuda_pull_array(l.biases_gpu, l.biases, l.n);
    cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
    cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_pull_array(l.scales_gpu, l.scales, l.n);
        cuda_pull_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_pull_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

void push_convolutional_layer(layer l)
{
    cuda_push_array(l.weights_gpu, l.weights, l.nweights);
#ifdef CUDNN_HALF
    cuda_convert_f32_to_f16(l.weights_gpu, l.nweights, l.weights_gpu16);
#endif
    cuda_push_array(l.biases_gpu, l.biases, l.n);
    cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
    cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_push_array(l.scales_gpu, l.scales, l.n);
        cuda_push_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_push_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

void update_convolutional_layer_gpu(layer l, update_args a)
{
    float learning_rate = a.learning_rate*l.learning_rate_scale;
    float momentum = a.momentum;
    float decay = a.decay;
    int batch = a.batch;

    if(a.adam){
        adam_update_gpu(l.weights_gpu, l.weight_updates_gpu, l.m_gpu, l.v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.nweights, batch, a.t);
        adam_update_gpu(l.biases_gpu, l.bias_updates_gpu, l.bias_m_gpu, l.bias_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        if(l.scales_gpu){
            adam_update_gpu(l.scales_gpu, l.scale_updates_gpu, l.scale_m_gpu, l.scale_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        }
    }else{
        // update weights:
        // weights_gpu = weights_gpu*(1 - decay*lr) + weight_updates_gpu*lr / (batch*subdivision) =
        //  weights_gpu*(1 - 0.0005*0.001) + weight_updates_gpu*0.001/(64*8) = 
        //  weights_gpu * 0.999 999 5 + weight_updates_gpu * 0.000 001 953125
        // 
        // weight_updates_gpu = (weight_updates_gpu - weights_gpu*decay*batch*subdivision)*momentum = 
        //  (weight_updates_gpu - weights_gpu * 0.0005 * 64 * 8) * 0.9 = 
        //  weight_updates_gpu*0.9 - weights_gpu*0.2304
        axpy_gpu(l.nweights, -decay*batch, l.weights_gpu, 1, l.weight_updates_gpu, 1);
        axpy_gpu(l.nweights, learning_rate/batch, l.weight_updates_gpu, 1, l.weights_gpu, 1);
        scal_gpu(l.nweights, momentum, l.weight_updates_gpu, 1);

        axpy_gpu(l.n, learning_rate/batch, l.bias_updates_gpu, 1, l.biases_gpu, 1);
        scal_gpu(l.n, momentum, l.bias_updates_gpu, 1);

        if(l.scales_gpu){
            axpy_gpu(l.n, learning_rate/batch, l.scale_updates_gpu, 1, l.scales_gpu, 1);
            scal_gpu(l.n, momentum, l.scale_updates_gpu, 1);
        }
    }
    if(l.clip){
        constrain_gpu(l.nweights, l.clip, l.weights_gpu, 1);
    }
}


